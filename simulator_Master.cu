#include "hip/hip_runtime.h"
#include "simulator_Master.cuh"
#include "functions_library.cuh"

simulator_Master::simulator_Master(string parameter_Master_Location)
{
    cout << "Intializing Simulator (based on CATE's architecture)\n";

    parameter_load Parameters = parameter_load();
    functions_library function = functions_library();

    vector<string> parameters_List = {
        "\"CUDA Device ID\"",
        "\"CPU cores\"",
        "\"GPU max units\"",
        "\"Intermediate folders\"",
        "\"Output folders\"",
        "\"Multi read\"",
        "\"Network profile\""};

    vector<string> found_Parameters = Parameters.get_parameters(parameter_Master_Location, parameters_List);

    cout << "Configuring folders:\n";

    output_Folder_location = Parameters.get_STRING(found_Parameters[4]);
    intermediate_Folder_location = Parameters.get_STRING(found_Parameters[3]);

    function.config_Folder(intermediate_Folder_location, "Intermediate");
    function.config_Folder(output_Folder_location, "Output");

    cout << "\nConfiguring hardware resources:\n\n";
    this->CPU_cores = Parameters.get_INT(found_Parameters[1]);
    cout << "Available CPU cores: " << this->CPU_cores << endl
         << endl;

    this->multi_Read = Parameters.get_STRING(found_Parameters[5]);
    cout << "Multiple read and write: " << this->multi_Read << endl
         << endl;

    this->CUDA_device_number = Parameters.get_INT(found_Parameters[0]);
    function.print_Cuda_device(this->CUDA_device_number, this->tot_Blocks, this->tot_ThreadsperBlock);

    this->gpu_Limit = Parameters.get_INT(found_Parameters[2]);

    cout << "Per round GPU max unit: " << this->gpu_Limit << endl
         << endl;

    configure_Network_Profile(Parameters.get_STRING(found_Parameters[6]), Parameters);
    cout << "\n";
}

void simulator_Master::configure_Network_Profile(string network_Profile_File, parameter_load &Parameters)
{
    cout << "Configuring network profile: " << network_Profile_File << endl;

    vector<string> parameters_List = {"\"Network type\""};
    vector<string> found_Parameters = Parameters.get_parameters(network_Profile_File, parameters_List);

    transform(found_Parameters[0].begin(), found_Parameters[0].end(), found_Parameters[0].begin(), ::toupper);

    parameters_List.clear();

    if (Parameters.get_STRING(found_Parameters[0]) == "BA MODEL")
    {
        cout << "\nBarabsi Albert model selected: \n";
        network_Model = "BA";

        parameters_List = {"\"BA model number of nodes\"",
                           "\"BA model standard new connections\""};
        found_Parameters = Parameters.get_parameters(network_Profile_File, parameters_List);

        number_of_Nodes_BA = Parameters.get_INT(found_Parameters[0]);
        cout << "Number of nodes: " << number_of_Nodes_BA << endl;
        connection_Model = Parameters.get_STRING(found_Parameters[1]);
        transform(connection_Model.begin(), connection_Model.end(), connection_Model.begin(), ::toupper);

        cout << "Node connection type: " << connection_Model << endl;

        if (connection_Model == "FIXED")
        {
            parameters_List = {"\"BA model fixed new connections\""};
            found_Parameters = Parameters.get_parameters(network_Profile_File, parameters_List);

            BA_FIXED = Parameters.get_INT(found_Parameters[0]);
            cout << "Fixed new connections: " << BA_FIXED << endl;
        }
        else if (connection_Model == "NEGATIVE BINOMIAL")
        {
            parameters_List = {"\"BA model Negative binomial sucesses\"",
                               "\"BA model Negative binomial probability\""};
            found_Parameters = Parameters.get_parameters(network_Profile_File, parameters_List);

            BA_NB_sucesses = Parameters.get_INT(found_Parameters[0]);
            BA_NB_probability = Parameters.get_FLOAT(found_Parameters[1]);

            cout << "Negative Binomial sucesses: " << BA_NB_sucesses << endl;
            cout << "Negative Binomial probability: " << BA_NB_probability << endl;
        }
        else if (connection_Model == "POISSON")
        {
            parameters_List = {"\"BA model Poisson mean\""};
            found_Parameters = Parameters.get_parameters(network_Profile_File, parameters_List);

            BA_Poisson_mean = Parameters.get_FLOAT(found_Parameters[0]);

            cout << "Poisson mean: " << BA_Poisson_mean << endl;
        }
    }
    else
    {
        cout << "ERROR Incorrect network selected. Please check \"Network type\" in the network parameter file: \"" << network_Profile_File << "\"";
        exit(-1);
    }
}

void simulator_Master::ingress()
{
    cout << "STEP 1: Configuring population network\n\n";

    string network_Summary_Location, network_node_Location;

    // ! Compatible for both BA and Caveman
    // INT, INT = Cave_ID and Node, for BA CaveID is 0 for all.
    vector<vector<int, int>> each_Nodes_Connection;

    if (network_Model == "BA")
    {
        BA_Model_Engine();
    }
    else
    {
        cout << "ERROR Incorrect network selected. Please check \"Network type\" in the network parameter file.\n";
        exit(-1);
    }
}

void simulator_Master::BA_Model_Engine()
{
    cout << "Intializing Barbasi Albert model network engine\n";
}
